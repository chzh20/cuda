#include"hip/hip_runtime.h"

#include <climits>
#include <cstddef>
#include<iostream>
#include <limits>
#include <stdexcept>
#include <type_traits>
#include<vector>
#include<string>
#include<random>
#include<functional>
#include<tuple>
#include"hipblas.h"
#include<hip/hip_fp16.h>
template <typename T>
void check(T result, const char *function, const char *file, size_t line)
{
    if (result)
    {
        std::cerr << "CUDA error at " << file << ":" << line << " function " << function
                  << " error code: " << hipGetErrorName(result)
                  << " error string: " << hipGetErrorString(result) << std::endl;
        // Optionally, you might want to reset the CUDA error state
        // cudaGetLastError(); // To reset the error state
        exit(EXIT_FAILURE); // EXIT_FAILURE is more standard than 1
    }
}

#define CUDACHECK(val) do { check((val), #val, __FILE__, __LINE__); } while (0)

class CudaTimer 

{
private:
    hipEvent_t start, stop;
    std::string m_kernalName;

public:
    // Constructor
    CudaTimer(const std::string& kernel_name = "") : m_kernalName(kernel_name){
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    // Destructor
    ~CudaTimer() {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    // Start timing
    void startTiming() {
        hipEventRecord(start, 0);
    }

    // Stop timing and return elapsed time in milliseconds
    float stopTiming() {
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        //std::cout<< m_kernalName << " elapsed time: " << milliseconds << " ms" << std::endl;
        return milliseconds;
        

    }
};


void printGPUInfo() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        std::cout << "Device " << i << ": " << deviceProp.name << std::endl;
        std::cout << "Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "Total Global Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "Shared Memory per Block: " << deviceProp.sharedMemPerBlock / 1024 << " KB" << std::endl;
        std::cout << "Max Threads per Block: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "Max Block Dimensions: (" << deviceProp.maxThreadsDim[0] << ", " << deviceProp.maxThreadsDim[1] << ", " << deviceProp.maxThreadsDim[2] << ")" << std::endl;
        std::cout << "Max Grid Dimensions: (" << deviceProp.maxGridSize[0] << ", " << deviceProp.maxGridSize[1] << ", " << deviceProp.maxGridSize[2] << ")" << std::endl;
        std::cout << std::endl;
    }
}



template <typename U = int>
static std::vector<U> generatevector(size_t N) {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_int_distribution<int> dis(-10000, 10000);
  std::vector<U> vec(N,0);
  for(int i = 0; i < N; i++){
    vec[i] = static_cast<U>(dis(gen));
  }
  return vec;
}

#define LOOP_TEST(test_func,n,baseline_time) \
{\
    float elapsed_time = 0.0f;\
    for(int i=0;i<n;++i)\
    {\
        elapsed_time += test_func();\
    }\
    std::cout<<#test_func<<" Average elapsed time: "<<elapsed_time/n<<" ms"<<std::endl;\
    if(baseline_time > 0)\
    {\
        std::cout<<"Speedup: "<<baseline_time/(elapsed_time/n)<<std::endl;\
    }\
}

template<typename T>
void gelu_cpu(T* odata,T* idata,size_t N)
{
    for(size_t i = 0;i<N;++i)
    {
        //gelu(x)= 0.5*x*(1+tanh(sqrt(2/pi)(x+0.044715*x^3))
        const T x = idata[i];
        const T cdf = T(0.5) * (T(1.0) + tanh((T(0.797884) * (x + T(0.044715) * x * x * x))));
        odata[i] = x * cdf;
    }
}




template<typename T>
__global__ void gelu_kernel(T* odata, const T* idata, size_t N)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        const T x = idata[idx];
        const T cdf = T(0.5) * (T(1.0) + tanh((T(0.797884) * (x + T(0.044715) * x * x * x))));
        odata[idx] = x * cdf;
    }
}





template<typename T>
void gelu_gpu(T* odata, T* idata, size_t N)
{
    const int threadsPerBlock = 256;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    gelu_kernel<<<blocks, threadsPerBlock>>>(odata, idata, N);
    CUDACHECK(hipGetLastError());
    CUDACHECK(hipDeviceSynchronize());
}


template<typename T,size_t Size>
struct alignas(sizeof(T)*Size) AlignedVector
{
    T data[Size];
    __host__ __device__ inline const T& operator[](size_t i) const { return data[i]; }
    __host__ __device__ inline T& operator[](size_t i) { return data[i]; }

};

template<typename T>
struct GeluFunctor
{
    static constexpr T sqrt_2_over_pi = static_cast<T>(0.79788456080286541);
    static constexpr T a =  static_cast<T>(0.044715);

    __device__ GeluFunctor() = default;

    __device__ __forceinline__ T operator()(T x) const
    {
       const T cdf = T(0.5) * (T(1.0) + tanh(sqrt_2_over_pi * (x + a * x * x * x)));
        return x * cdf;
    }
};

__device__ __forceinline__ float TanhApprox(float x)
{
#if __CUDA_ARCH__ >= 800
    float r;
    asm("tanh.approx.f32 %0,%1;" : "=f"(r) : "f"(x));
    return r;
#else
    return tanhf(x);
#endif

}


template<>
struct GeluFunctor<half>
{
    static constexpr float sqrt_2_over_pi= 0.79788456080286541f;
    static constexpr float a = 0.044715;
   
    __device__ GeluFunctor() = default;


    __device__ __forceinline__ half operator()(half x) const
    {
#if __CUDA_ARCH__ >= 800
        const float tanh_in = __half2float(__float2half_rn(sqrt_2_over_pi) * (x + __float2half_rn(a) * x * x * x));
        const float tanh_out = TanhApprox(tanh_in);
        const half cdf = 0.5h * (1.0h + __float2half_rn(tanh_out));
        return x * cdf;

#else
     GeluFunctor <float> float_gelu;
     return static_cast<half>(float_gelu(static_cast<float>(x)));
#endif

    }
};


template<size_t VecSize>
__global__ void FP16GeluKernel(half* odata,const half* idata,size_t N)
{
    //向量化 load & store
    int offset = (blockIdx.x * blockDim.x + threadIdx.x) * VecSize;
    int stride = blockDim.x * gridDim.x * VecSize;
    GeluFunctor<half> gelu;
    __half y_reg[VecSize];

    using ArrT = AlignedVector<__half, VecSize>;
    for(; offset < N; offset += stride)
    {
        const __half* in = idata + offset;
        __half* out = odata + offset;

        if(VecSize == 1)
        {
            y_reg[0] = gelu(in[0]);
        }
        else
        {
            #pragma unroll
            for(int i = 0;i < VecSize;++i)
            {
                y_reg[i] = gelu(in[i]);
            }
        }

        *reinterpret_cast<ArrT*>(out) = *reinterpret_cast<ArrT*>(y_reg);
    }
}


template<typename T>
float test_gelu()
{
    const size_t N = 1 << 20;  // 大约100万个元素
    std::vector<T> h_input = generatevector<T>(N);
    std::vector<T> h_output_cpu(N);
    std::vector<T> h_output_gpu(N);

    // 分配GPU内存
    T *d_input, *d_output;
    CUDACHECK(hipMalloc(&d_input, N * sizeof(T)));
    CUDACHECK(hipMalloc(&d_output, N * sizeof(T)));

    // 将输入数据复制到GPU
    CUDACHECK(hipMemcpy(d_input, h_input.data(), N * sizeof(T), hipMemcpyHostToDevice));

    // CPU计算
    gelu_cpu(h_output_cpu.data(), h_input.data(), N);

    // GPU计算
    CudaTimer timer("GELU GPU");
    timer.startTiming();
    gelu_gpu(d_output, d_input, N);
    float elapsed_time = timer.stopTiming();

    // 将结果复制回主机
    CUDACHECK(hipMemcpy(h_output_gpu.data(), d_output, N * sizeof(T), hipMemcpyDeviceToHost));

    // 验证结果
    double max_error = 0.0;
    for (size_t i = 0; i < N; ++i) {
        double error = std::abs(static_cast<double>(h_output_cpu[i]) - static_cast<double>(h_output_gpu[i]));
        max_error = std::max(max_error, error);
    }
    std::cout << "最大误差: " << max_error << std::endl;

    // 释放GPU内存
    CUDACHECK(hipFree(d_input));
    CUDACHECK(hipFree(d_output));

    return elapsed_time;
}



void looptest()
{
    std::cout << "测试单精度GELU:" << std::endl;
    LOOP_TEST(test_gelu<float>, 10, 1);

    std::cout << "\n测试双精度GELU:" << std::endl;
    LOOP_TEST(test_gelu<double>, 10, 1);
}



int main()
{
    printGPUInfo();
    looptest();
    return 0;
}


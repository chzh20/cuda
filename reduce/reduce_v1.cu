#include"hip/hip_runtime.h"

#include <iostream>
#include <vector>
#include <chrono>
#include <random>
#include<execution>
#include <algorithm>
#include<numeric>

/*

reduce_basic 存在以下问题：
1. 取模操作非常耗时
2. if(tid % (2 * stride) == 0) 导致了大量的control divergence，即线程分支非常多，导致了大量的warp divergence
3. 没有使用shared memory，导致了大量的global memory访问，global memory访问非常慢
*/

template<typename T, typename OP>
__global__ void reduce_basic(T * input,T* output, int n, OP op)
{
    int tid = threadIdx.x;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= n)
    {
        return;
    }
    T* data = input + blockIdx.x * blockDim.x;
    for(int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if(tid % (2 * stride) == 0)
        {
            data[tid] = op(data[tid], data[tid + stride]);
        }
        __syncthreads();
    }
    if(tid == 0)
    {
        output[blockIdx.x] = data[0];
    }

}





/*
reduce_v1 有以下优点：
1.使用shared memory，减少了global memory访问
2.通过 int index = 2 * stride * tid; 较少了取模操作和control divergence
reduce_v1每个warp中都有control divergence；而redece_v2中只有第一个warp有control divergence(当线程数小于32)

有以下缺点：
1. 仍然存在大量的warp divergence.
2. 有一半的线程没有工作，浪费了计算资源.
假设blocksize=256,则分配256/32=8个warp.只有前4个warp有工作，后4个warp没有工作.


reduce1的最大问题是bank冲突。我们把目光聚焦在这个for循环中。并且只聚焦在0号warp。
在第一次迭代中，0号线程需要去load shared memory的0号地址以及1号地址的数，然后写回到0号地址。
而此时，这个warp中的16号线程，需要去load shared memory中的32号地址和33号地址。
可以发现，0号地址跟32号地址产生了2路的bank冲突。
在第2次迭代中，0号线程需要去load shared memory中的0号地址和2号地址。
这个warp中的8号线程需要load shared memory中的32号地址以及34号地址，
16号线程需要load shared memory中的64号地址和68号地址，24号线程需要load shared memory中的96号地址和100号地址。
又因为0、32、64、96号地址对应着同一个bank，所以此时产生了4路的bank冲突。现在，可以继续算下去，
8路bank冲突，16路bank冲突。由于bank冲突，所以reduce1性能受限。下图说明了在load第一个数据时所产生的bank冲突。
*/

template<typename T, int BlockSize, typename OP>
__global__ void reduce_v1(T * input,T* output, int n, OP op)
{
    int tid = threadIdx.x;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= n)
    {
        return;
    }
    T* data = input + blockIdx.x * blockDim.x;
    __shared__  T smem[BlockSize];
    smem[tid] = data[tid];
    __syncthreads();

    for(int stride = 1; stride < blockDim.x; stride *= 2)
    {
        int index = 2 * stride * tid;
        if(index < blockDim.x ) // 有一半的线程没有工作
        {
            smem[tid] = op(smem[index], smem[index + stride]);
        }
        __syncthreads();
    }
    if(tid == 0)
    {
        output[blockIdx.x] = smem[0];
    }
}


/*

reduce_v2 有以下优点：
1. 通过调整交错访问，反转了元素的步幅：步幅从线程块大小的一半开始，然后在每次迭代中减半
每个线程在每轮中添加两个由当前步幅分隔的元素，以生成部分和。
2.减少了warp divergence：假设blocksize=256,则分配256/32=8个warp.
在第一轮中,warp0和warp1有工作，warp2和warp3没有工作
在第二轮中，warp0有工作，warp1没有工作，warp2和warp3有工作
在第三轮中，warp0和warp1没有工作，warp2有工作，warp3没有工作
2. 通过调整交错访问，减少了global memory访问

缺点：
1. stride =  blockDim.x/2： 有一半的线程没有工作，浪费了计算资源.

*/

template<typename T, int BlockSize, typename OP>
__global__ void reduce_v2(T * input,T* output, int n, OP op)
{
    int tid = threadIdx.x;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= n)
    {
        return;
    }
    T* data = input + blockIdx.x * blockDim.x;
    __shared__  T smem[BlockSize];
    smem[tid] = data[tid];
    __syncthreads();

    for(int stride =  blockDim.x/2; stride > 0; stride >>= 1)
    {
        if( tid < stride )
        {
            smem[tid] = op(smem[tid], smem[tid + stride]);
        }
        __syncthreads();
    }
    if(tid == 0)
    {
        output[blockIdx.x] = smem[0];
    }
}


/* 

reduce_v3 有以下优点：
1.一次处理2个block的数据. 

*/

template <typename T, int BlockSize, typename OP>
__global__ void reduce_v3(T *input, T *output, int n, OP op) {
  int tid = threadIdx.x;
  int idx = threadIdx.x + blockIdx.x * (blockDim.x * 2);
  if (idx >= n)
  {
      return;
  }
  T *data = input + blockIdx.x * (blockDim.x * 2);
  __shared__ T smem[BlockSize];
  if(tid + blockDim.x < n) {
    smem[tid] = op(data[tid], data[tid + blockDim.x]);
  }
  __syncthreads();

  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      smem[tid] = op(smem[tid], smem[tid + stride]);
    }
    __syncthreads();
  }
  if (tid == 0) {
    output[blockIdx.x] = smem[0];
  }
}
/*
reduce_v4 有以下优点：
当stride<=32时，只有第一个warp在工作
warp内，指令是SIMD同步, 在单个 warp 内，线程的执行是 SIMD 同步的，这意味着同一 warp 内的线程以相同的步调执行相同的指令。
因此，在 warp 内进行归约时，不需要额外的同步.
**Volatile 关键字**：`volatile` 关键字确保编译器不会对共享内存的访问进行优化，保证每次都从内存中读取最新的值。
因此可以把最后6次迭代展开
*/


template <typename T, int BlockSize, typename OP>
__global__ void reduce_v4( T *input, T *output, int n, OP op) {
  int tid = threadIdx.x;
  int idx = threadIdx.x + blockIdx.x * (blockDim.x * 2);
  if(idx>n) return;
  __shared__ T smem[BlockSize];

  smem[tid] =input[idx];
  if (idx + blockDim.x < n)
  {
      smem[tid] = op(smem[tid], input[idx + blockDim.x]);
  }
  __syncthreads();

  for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
    if (tid < stride) {
      smem[tid] = op(smem[tid], smem[tid + stride]);
    }
    __syncthreads();
  }
  //unroll the last 6 iterations
  if (tid < 32) {
    volatile T *vsmem = smem;
    vsmem[tid] = op(vsmem[tid], vsmem[tid + 32]);
    vsmem[tid] = op(vsmem[tid], vsmem[tid + 16]);
    vsmem[tid] = op(vsmem[tid], vsmem[tid + 8]);
    vsmem[tid] = op(vsmem[tid], vsmem[tid + 4]);
    vsmem[tid] = op(vsmem[tid], vsmem[tid + 2]);
    vsmem[tid] = op(vsmem[tid], vsmem[tid + 1]);
  }

  if (tid == 0) {
    output[blockIdx.x] = smem[0];
  }
}


template<typename T,typename OP>
__device__ __forceinline__ T warpshulf(T val, OP op)
{
    for(int offset = warpSize/2; offset > 0; offset /= 2)
    {
        val = op(val, __shfl_down_sync(0xffffffff, val, offset));
    }
    return val;
}

template<int BLOCK_SIZE,typename T,typename OP>
__global__ void reduce_v5(T* input, T* output, int n, OP op)
{
    int tid = threadIdx.x;
    int idx = threadIdx.x + blockIdx.x * blockDim.x*2;
    int gridSize = BLOCK_SIZE * 2 * gridDim.x;
    T* data = input + blockIdx.x * BLOCK_SIZE * 2;
    T sum = data[tid];
    while(idx < n)
    {
        sum = op(sum, data[tid + BLOCK_SIZE]);
        idx += gridSize;
    }

    constexpr int warps = BLOCK_SIZE / 32;
    static __shared__ T smem[warps];
    sum = warpshulf(sum, op);

    int lane = tid % warpSize;
    int wid = tid / warpSize;
    if(lane == 0) smem[wid] = sum;
    __syncthreads();
    
    sum = (tid < warps) ? smem[tid] : 0;
    if(wid == 0)
    {
        sum = warpshulf(sum, op);
    }
    if(tid == 0)
    {
        output[blockIdx.x] = sum;
    }
    
}



// Simple functor for summation
struct AddOp {
    __device__ __forceinline__ float operator()(float a, float b) const {
        return a + b;
    }
};

// Utility to check CUDA errors
static inline void checkCuda(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << " : " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
bool checkResult(const std::vector<float>& hOut,float hOutRef) {
    // for (int i = 0; i < hOut.size(); i++) {
    //     if (hOut[i] != hOutRef[i]) {
    //         std::cerr << "Mismatch at index " << i << ": " << hOut[i] << " != " << hOutRef[i] << std::endl;
    //         return false;
    //     }
    // }
    float sum = std::accumulate(hOut.begin(), hOut.end(), 0.f);
    if (std::abs(sum - hOutRef) > 1e-5) {
        std::cerr << "Mismatch in sum: " << sum << " != " << hOutRef << std::endl;
        return false;
    }
    return true;
}

void benchmarkKernels(int n, int blockSize) {
    // Host memory
    std::vector<float> hIn(n), hOut( (n + blockSize - 1) / blockSize );
    std::mt19937 gen(0);
    std::uniform_real_distribution<float> dist(0.f, 1.f);
    for(int i = 0; i < n; i++)
        hIn[i] = dist(gen);

    // Reference computation
    auto timeStart = std::chrono::high_resolution_clock::now();
    auto hOutRef = std::reduce(std::execution::par, hIn.begin(), hIn.end(), 0.f, std::plus<float>());
    auto timeEnd = std::chrono::high_resolution_clock::now();
    std::cout << "Reference sum: " << hOutRef << " took "
              << std::chrono::duration_cast<std::chrono::milliseconds>(timeEnd - timeStart).count()
              << " ms" << std::endl;

    // Device memory
    float *dIn = nullptr, *dOut = nullptr;
    checkCuda(hipMalloc(&dIn, n * sizeof(float)), "Failed to allocate dIn");
    checkCuda(hipMalloc(&dOut, hOut.size() * sizeof(float)), "Failed to allocate dOut");
    checkCuda(hipMemcpy(dIn, hIn.data(), n * sizeof(float), hipMemcpyHostToDevice),
              "Failed to copy to dIn");

    // Helper lambda for timing
    auto timeKernel = [&](auto kernelLaunch, const char* label) {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        kernelLaunch();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms = 0.f;
        hipEventElapsedTime(&ms, start, stop);
        std::cout << label << " took " << ms << " ms" << std::endl;
        hipEventDestroy(start);
        hipEventDestroy(stop);
    };
    auto checkKernalResult = [&](const char * label) {
        hipMemcpy(hOut.data(), dOut, hOut.size() * sizeof(float), hipMemcpyDeviceToHost);
        if(!checkResult(hOut, hOutRef)) {
            std::cerr << "Mismatch in " << label << std::endl;
        }
        hOut.clear();
    };

    // Grid size
    int gridSize = (n + blockSize - 1) / blockSize;

    // Launch each kernel (names assumed from existing code)
    timeKernel([&](){
        reduce_basic<float, AddOp><<<gridSize, blockSize>>>(dIn, dOut, n, AddOp());
        checkCuda(hipGetLastError(), "Failed to launch reduce_basic");
        hipDeviceSynchronize();
    }, "reduce_basic");
    checkKernalResult("reduce_basic");

    timeKernel([&](){
        reduce_v1<float, 256, AddOp><<<gridSize, 256>>>(dIn, dOut, n, AddOp());
        checkCuda(hipGetLastError(), "Failed to launch reduce_v1");
        hipDeviceSynchronize();
    }, "reduce_v1");
    checkKernalResult("reduce_v1");

    timeKernel([&](){
        reduce_v2<float, 256, AddOp><<<gridSize, 256>>>(dIn, dOut, n, AddOp());
        checkCuda(hipGetLastError(), "Failed to launch reduce_v2");
        hipDeviceSynchronize();
    }, "reduce_v2");
    checkKernalResult("reduce_v2");

    timeKernel([&](){
        reduce_v3<float, 256, AddOp><<<gridSize/2, 256>>>(dIn, dOut, n, AddOp());
        checkCuda(hipGetLastError(), "Failed to launch reduce_v3");
        hipDeviceSynchronize();
    }, "reduce_v3");
    checkKernalResult("reduce_v3");

    // timeKernel([&](){
    //     dim3 gridDim(gridSize / 2);
    //     dim3 blockDim(256);
    //     reduce_v4<float, 256, AddOp><<<gridDim, blockDim>>>(dIn, dOut, n, AddOp());
    //     checkCuda(cudaGetLastError(), "Failed to launch reduce_v4");
    //     cudaDeviceSynchronize();
    // }, "reduce_v4");
    // cudaMemcpy(hOut.data(), dOut, hOut.size() * sizeof(float), cudaMemcpyDeviceToHost);
    // if(!checkResult(hOut, hOutRef)) {
    //     std::cerr << "Mismatch in reduce_v4" << std::endl;
    // }


    timeKernel([&](){
        reduce_v5<256, float, AddOp><<<gridSize, 256>>>(dIn, dOut, n, AddOp());
        checkCuda(hipGetLastError(), "Failed to launch reduce_v5");
        hipDeviceSynchronize();
    }, "reduce_v5");
    checkKernalResult("reduce_v5");



    // Cleanup
    hipFree(dIn);
    hipFree(dOut);
}

int main() {
    int n =32*1024*1024;
    int blockSize = 256;
    benchmarkKernels(n, blockSize);
    return 0;
}


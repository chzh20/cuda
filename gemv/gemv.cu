#include"hip/hip_runtime.h"

#include <__clang_cuda_builtin_vars.h>
#include <climits>
#include <cstddef>
#include<iostream>
#include <limits>
#include <stdexcept>
#include <type_traits>
#include<vector>
#include<string>
#include<random>
#include<functional>
#include<tuple>
#include"hipblas.h"
#include<hip/hip_fp16.h>
template <typename T>
void check(T result, const char *function, const char *file, size_t line)
{
    if (result)
    {
        std::cerr << "CUDA error at " << file << ":" << line << " function " << function
                  << " error code: " << hipGetErrorName(result)
                  << " error string: " << hipGetErrorString(result) << std::endl;
        // Optionally, you might want to reset the CUDA error state
        // cudaGetLastError(); // To reset the error state
        exit(EXIT_FAILURE); // EXIT_FAILURE is more standard than 1
    }
}

#define CUDACHECK(val) do { check((val), #val, __FILE__, __LINE__); } while (0)



template<typename T>
void gemvCPU(const T* mat,const T*vec,T*res,int m,int n)
{
    for(int i =0; i<m;++i)
    {
        res[i] = 0;
        for(int j =0; j<n;++j)
        {
            res[i] += mat[i*n+j]*vec[j];
        }
    }
}
template<typename T>
bool checkGroundTruth(const T* res1,const T* res2,int m)
{
    for(int i =0; i<m;++i)
    {
        if(fabs(res1[i]-res2[i])>1e-3)
        {
            std::cerr<<"Error at "<<i<<" "<<res1[i]<<" "<<res2[i]<<std::endl;
            return false;
        }
    }
    return true;
}


template<typename T>
struct Vec{
    static constexpr size_t size =4;
};
template<>
struct Vec<half2>{
    static constexpr size_t size = 8;
};


template<template<typename> typename ReductionOp,typename T>
__device__ __forceinline__ T warpReduce(T val)
{
    for(int offset = warpSize/2; offset>0;offset/=2)
    {
        val = ReductionOp<T>()(val,__shfl_down_sync(0xffffffff,val,offset));
    }
    return val;
}

template<template<typename> typename ReductionOp,typename T>
__device__ __forceinline__ T blockReduce(T val)
{
    static __shared__ T shared[64]; //warpsize 2048/32 = 64
    int tid = threadIdx.x;
    int lane = threadIdx.x%warpSize;
    int wid = threadIdx.x/warpSize;
    int  warp_nums = (blockDim.x+warpSize-1)/warpSize;
    val = warpReduce<ReductionOp,T>(val);
    if(lane == 0)
    {
        shared[wid] = val;
    }
    __syncthreads();
    T wrap_val = (tid<warp_nums)?shared[tid]:T(0);
    return  warpReduce<ReductionOp,T>(wrap_val);
}


template<typename T>
struct SumOp
{
    __device__ __forceinline__ T operator()(const T &a,const T &b)
    {
        return a+b;
    }
};
template<>
struct SumOp<half>
{
    __device__ __forceinline__ half operator()(const half &a,const half &b)
    {
        return __hadd(a,b);
    }
};



//compute one element per block
//m blocks
template<size_t VECS_PER_THREAD,size_t VEC_SIZE>
__global__ void gemvKernel(float* mat,float* vec,float* dst,int m,int n)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x; // row index
    float thread_local_sum = 0.0f;
    for(int i =0; i< VECS_PER_THREAD;++i)
    {
        float4 * mat4 = reinterpret_cast<float4*>(&mat[bid*n+ tid*VEC_SIZE]);
        float4 * vec4 = reinterpret_cast<float4*>(&vec[tid*VEC_SIZE]);
        thread_local_sum += mat4[i].x*vec4[i].x+mat4[i].y*vec4[i].y+mat4[i].z*vec4[i].z+mat4[i].w*vec4[i].w;
    }
    float block_sum = blockReduce<SumOp,float>(thread_local_sum);
    if(tid == 0)
    {
        dst[bid] = block_sum;
    }
    __syncthreads();

}

template<size_t VECS_PER_THREAD,size_t VEC_SIZE>
__global__ void gemvKernel(half* mat,half* vec,half* dst,int m,int n)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    half thread_local_sum = 0.0f;
    for(int i = 0; i< VECS_PER_THREAD; ++i)
    {
       float4 * mat4 = reinterpret_cast<float4*>(&mat[bid*n+tid*VEC_SIZE]);
       float4 * vec4 = reinterpret_cast<float4*>(&vec[tid*VEC_SIZE]);
       half2*   vec_h1 =(half2*)&vec4[i].x;
       half2*   vec_h2 =(half2*)&vec4[i].y;
       half2*   vec_h3 = (half2*)&vec4[i].z;
       half2*   vec_h4 = (half2*)&vec4[i].w;

       half2*   mat_h1 = (half2*)&mat4[i].x;
       half2*   mat_h2 = (half2*)&mat4[i].y;
       half2*   mat_h3 = (half2*)&mat4[i].z;
       half2*   mat_h4 = (half2*)&mat4[i].w;
       half2 res1 = __hmul2(*mat_h1,*vec_h1);
       half2 res2 = __hmul2(*mat_h2,*vec_h2);
       half2 res3 =__hmul2(*mat_h3,*vec_h3);
       half2 res4 = __hmul2(*mat_h4,*vec_h4);
       half2 res = __hadd2(__hadd2(res1, res2), __hadd2(res3, res4));
       thread_local_sum = __hadd(reinterpret_cast<half&>(res.x),reinterpret_cast<half&>(res.y));
    }
    half block_sum = blockReduce<SumOp,half>(thread_local_sum);
    if(tid ==0)
    {
        dst[bid] = block_sum;
    }
    __syncthreads();
}



//VEC_SIZE表示每个向量的大小
//VECS_PER_THREAD表示每个线程处理的向量数
//THREAD_NUMS表示每个block的线程数
template<size_t VECS_PER_THREAD, size_t VEC_SIZE,size_t THREAD_NUMS>
struct DispatchLauncher
{
    template<typename T>
    static void launch(const T* mat,const T* vec,T* dst,int m,int n)
    {
        dim3 grid(m);
        dim3 block(THREAD_NUMS);
        float time=0.0f;
        hipEvent_t start,stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start,0);
        gemvKernel<T><<<grid,block>>>(mat,vec,dst,m,n);
        CUDACHECK(hipGetLastError());
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time,start,stop);
        std::cout<<"Time: "<<time<<std::endl;
    }
};






template<typename T>
void gemv_kernel(T* mat,T*d_mat,T*vec,T*d_vec,T*dst,T*d_dst)
{
    constexpr size_t M = 256;
    constexpr size_t N = 2048;
    vec = (T*)malloc(N*sizeof(T));
    mat = (T*)malloc(M*N*sizeof(T));
    dst = (T*)malloc(M*sizeof(T));

    hipMalloc(&d_mat,M*N*sizeof(T));
    hipMalloc(&d_vec,N*sizeof(T));
    hipMalloc(&d_dst,M*sizeof(T));

    for(int i=0;i<N;++i)
    {
        vec[i] = (T)(rand()%100)/100;
    }
    for(int i=0;i<M*N;++i)
    {
        mat[i] = (T)(rand()%100)/100;
    }

    hipMemcpy(d_mat,mat,M*N*sizeof(T),hipMemcpyHostToDevice);
    hipMemcpy(d_vec,vec,N*sizeof(T),hipMemcpyHostToDevice);

    
    constexpr size_t THREAD_NUMS = 256;
    constexpr size_t VEC_SIZE = Vec<T>::size;
    constexpr size_t VECS_PER_THREAD = (N/THREAD_NUMS)/VEC_SIZE;

    DispatchLauncher<VECS_PER_THREAD,VEC_SIZE,THREAD_NUMS>::template launch(d_mat,d_vec,d_dst,M,N);

    hipMemcpy(dst,d_dst,M*sizeof(T),hipMemcpyDeviceToHost);

    T* dst_cpu = (T*)malloc(M*sizeof(T));
    gemvCPU(mat,vec,dst_cpu,M,N);
    if(!checkGroundTruth(dst,dst_cpu,M))
    {
        std::cerr<<"Error"<<std::endl;
    }
    else
    {
        std::cout<<"Success"<<std::endl;
    }
    free(vec);
    free(mat);
    free(dst);
    free(dst_cpu);
    hipFree(d_mat);
    hipFree(d_vec);
    hipFree(d_dst);


}
int main()
{
    return 0;
}